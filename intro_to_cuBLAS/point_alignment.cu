/**
 * CUDA Point Alignment
 * George Stathopoulos, Jenny Lee, Mary Giambrone, 2019*/ 

#include <cstdio>
#include <stdio.h>
#include <fstream>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>

#include "hip/hip_runtime_api.h"
#include <string>
#include <fstream>

#include "obj_structures.h"

// hip/hip_runtime_api.h contains the error checking macros. note that they're called
// CUDA_CALL, CUBLAS_CALL, and CUSOLVER_CALL instead of the previous names

#define IDX2C(i,j,ld) (((j)*(ld))+(i)) // i is row, j is col


// custom helper to proint matrices (must be stored on CPU)
void printMatrix(std::string title, float *matrix, int rows, int cols) {
    std::cout << title << "\n";
    for (int r = 0; r < rows; r++) {
        for (int c = 0; c < cols; c++) {
            std::cout << matrix[IDX2C(r, c, cols)] << " ";
        }
        std::cout << "\n";
    }
}


int main(int argc, char *argv[]) {

    if (argc != 4)
    {
        printf("Usage: ./point_alignment [file1.obj] [file2.obj] [output.obj]\n");
        return 1;
    }

    std::string filename, filename2, output_filename;
    filename = argv[1];
    filename2 = argv[2];
    output_filename = argv[3];

    std::cout << "Aligning " << filename << " with " << filename2 <<  std::endl;
    Object obj1 = read_obj_file(filename);
    std::cout << "Reading " << filename << ", which has " << obj1.vertices.size() << " vertices" << std::endl;
    Object obj2 = read_obj_file(filename2);

    std::cout << "Reading " << filename2 << ", which has " << obj2.vertices.size() << " vertices" << std::endl;
    if (obj1.vertices.size() != obj2.vertices.size())
    {
        printf("Error: number of vertices in the obj files do not match.\n");
        return 1;
    }

    ///////////////////////////////////////////////////////////////////////////
    // Loading in obj into vertex Array
    ///////////////////////////////////////////////////////////////////////////

    // Assume num of points match by this point (would have errored otherwise)
    int point_dim = 4; // 3 spatial + 1 homogeneous
    int num_points = obj1.vertices.size();

    // in col-major
    float * x1mat = vertex_array_from_obj(obj1);
    float * x2mat = vertex_array_from_obj(obj2);

    ///////////////////////////////////////////////////////////////////////////
    // Point Alignment
    ///////////////////////////////////////////////////////////////////////////

    // DONE: Initialize cublas handle
    hipblasHandle_t handle;
    CUBLAS_CALL(hipblasCreate(&handle));

    float *dev_x1mat;
    float *dev_x2mat;
    float *dev_xx4x4;
    float *dev_x1Tx2;

    // DONE: Allocate device memory and copy over the data onto the device
    // Hint: Use hipblasSetMatrix() for copying
    CUDA_CALL(hipMalloc((void **)&dev_x1mat, num_points*point_dim*sizeof(float)));
    CUBLAS_CALL(hipblasSetMatrix(num_points, point_dim, sizeof(float), 
        x1mat, num_points, dev_x1mat, num_points));

    CUDA_CALL(hipMalloc((void **)&dev_x2mat, num_points*point_dim*sizeof(float)));
    CUBLAS_CALL(hipblasSetMatrix(num_points, point_dim, sizeof(float), 
        x2mat, num_points, dev_x2mat, num_points));

    // Now, proceed with the computations necessary to solve for the linear
    // transformation.

    float one = 1;
    float zero = 0;

    // Dimension Notes:
    // x1 and x2 have dim (R,C) = Nx4
    // xx4xx4 and x1Tx2 have dim (R,C) = 4xN * Nx4 = 4x4

    // DONE: First calculate xx4x4 and x1Tx2
    // Following two calls should correspond to:
    //   xx4x4 = Transpose[x1mat] . x1mat
    //   x1Tx2 = Transpose[x1mat] . x2mat
    CUDA_CALL(hipMalloc((void **)&dev_xx4x4, point_dim*point_dim*sizeof(float)));
    CUBLAS_CALL(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
        point_dim, point_dim, num_points, &one,
        dev_x1mat, num_points, dev_x1mat, num_points, &zero, 
        dev_xx4x4, point_dim));

    CUDA_CALL(hipMalloc((void **)&dev_x1Tx2, point_dim*point_dim*sizeof(float)));
    CUBLAS_CALL(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
        point_dim, point_dim, num_points, &one,
        dev_x1mat, num_points, dev_x2mat, num_points, &zero, 
        dev_x1Tx2, point_dim));

    // DONE: Finally, solve the system using LU-factorization! We're solving
    //         xx4x4 . m4x4mat.T = x1Tx2   i.e.   m4x4mat.T = Inverse[xx4x4] . x1Tx2
    //
    //       Factorize xx4x4 into an L and U matrix, ie.  xx4x4 = LU
    //
    //       Then, solve the following two systems at once using cusolver's getrs
    //           L . temp  =  P . x1Tx2
    //       And then then,
    //           U . m4x4mat = temp
    //
    //       Generally, pre-factoring a matrix is a very good strategy when
    //       it is needed for repeated solves.

    // DONE: Make handle for cuSolver
    hipsolverHandle_t solver_handle;
    CUSOLVER_CALL(hipsolverDnCreate(&solver_handle));

    // DONE: Initialize work buffer using hipsolverDnSgetrf_bufferSize
    float * work;
    int Lwork;
    CUSOLVER_CALL(hipsolverDnSgetrf_bufferSize(solver_handle, 
        point_dim, point_dim, dev_xx4x4, point_dim, &Lwork));

    // DONE: Compute buffer size and prepare memory
    CUDA_CALL(hipMalloc((void **)&work, Lwork * sizeof(float)));

    // DONE: Initialize memory for pivot array, with a size of point_dim
    int * pivots;
    CUDA_CALL(hipMalloc((void **)&pivots, point_dim * sizeof(int)));

    int *info;
    CUDA_CALL(hipMalloc((void **)&info, sizeof(int)));

    // DONE: Now, call the factorizer hipsolverDnSgetrf, using the above initialized data
    CUSOLVER_CALL(hipsolverDnSgetrf(solver_handle, 
        point_dim, point_dim, dev_xx4x4, point_dim, work, pivots, info));

    // DONE: Finally, solve the factorized version using a direct call to hipsolverDnSgetrs
    CUSOLVER_CALL(hipsolverDnSgetrs(solver_handle, HIPBLAS_OP_N, 
        point_dim, point_dim, dev_xx4x4, point_dim, pivots, dev_x1Tx2, point_dim, info));
    
    // DONE: Destroy the cuSolver handle
    CUSOLVER_CALL(hipsolverDnDestroy(solver_handle));

    // DONE: Copy final transformation back to host. Note that at this point
    // the transformation matrix is transposed
    float * out_transformation = (float *)malloc(point_dim*point_dim*sizeof(float));
    CUBLAS_CALL(hipblasGetMatrix(point_dim, point_dim, sizeof(float), 
        dev_x1Tx2, point_dim, out_transformation, point_dim));

    // DONE: Don't forget to set the bottom row of the final transformation
    //       to [0,0,0,1] (right-most columns of the transposed matrix)
    out_transformation[IDX2C(0,3,point_dim)] = 0;
    out_transformation[IDX2C(1,3,point_dim)] = 0;
    out_transformation[IDX2C(2,3,point_dim)] = 0;
    out_transformation[IDX2C(3,3,point_dim)] = 1;

    // Print transformation in row order.
    printMatrix("Transformation Matrix", out_transformation, point_dim, point_dim);


    ///////////////////////////////////////////////////////////////////////////
    // Transform point and print output object file
    ///////////////////////////////////////////////////////////////////////////

    // DONE: Allocate and Initialize data matrix
    float * dev_pt;
    CUDA_CALL(hipMalloc((void **)&dev_pt, num_points*point_dim*sizeof(float)));
    CUBLAS_CALL(hipblasSetMatrix(num_points, point_dim, sizeof(float), 
        x1mat, num_points, dev_pt, num_points));

    // DONE: Allocate and Initialize transformation matrix
    float * dev_trans_mat;
    CUDA_CALL(hipMalloc((void **)&dev_trans_mat, point_dim*point_dim*sizeof(float)));
    CUBLAS_CALL(hipblasSetMatrix(point_dim, point_dim, sizeof(float), 
        out_transformation, point_dim, dev_trans_mat, point_dim));

    // DONE: Allocate and Initialize transformed points
    float * dev_trans_pt;
    CUDA_CALL(hipMalloc((void **)&dev_trans_pt, num_points*point_dim*sizeof(float)));

    float one_d = 1;
    float zero_d = 0;

    // DONE: Transform point matrix
    //          (4x4 trans_mat) . (nx4 pointzx matrix)^T = (4xn transformed points)
    CUBLAS_CALL(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
        point_dim, num_points, point_dim, &one_d,
        dev_trans_mat, point_dim, dev_pt, num_points, &zero_d, 
        dev_trans_pt, point_dim));

    // So now dev_trans_pt has shape (4 x n)
    // DONE: Copy final transformed vertices back over to the host
    float * trans_pt = (float *)malloc(num_points*point_dim*sizeof(float));
    CUBLAS_CALL(hipblasGetMatrix(point_dim, num_points, sizeof(float), 
        dev_trans_pt, point_dim, trans_pt, point_dim));

    // get Object from transformed vertex matrix
    Object trans_obj = obj_from_vertex_array(trans_pt, num_points, point_dim, obj1);

    // print Object to output file
    std::ofstream obj_file (output_filename);
    print_obj_data(trans_obj, obj_file);

    // free CPU memory
    free(trans_pt);

    ///////////////////////////////////////////////////////////////////////////
    // Free Memory
    ///////////////////////////////////////////////////////////////////////////

    // DONE: Destory cublas handle
    CUBLAS_CALL(hipblasDestroy(handle));

    // DONE: Free GPU memory
    CUDA_CALL(hipFree(dev_x1mat));
    CUDA_CALL(hipFree(dev_x2mat));
    CUDA_CALL(hipFree(dev_xx4x4));
    CUDA_CALL(hipFree(dev_x1Tx2));
    CUDA_CALL(hipFree(work));
    CUDA_CALL(hipFree(pivots));
    CUDA_CALL(hipFree(info));
    CUDA_CALL(hipFree(dev_trans_mat));
    CUDA_CALL(hipFree(dev_pt));
    CUDA_CALL(hipFree(dev_trans_pt));

    // DONE: Free CPU memory
    free(out_transformation);
    free(x1mat);
    free(x2mat);

}

